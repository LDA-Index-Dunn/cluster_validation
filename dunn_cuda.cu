#include "hip/hip_runtime.h"
// This is the CUDA program for Dunn index calculation
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>

// Test_k2_f2_10, Iris_k3_f4_150 - Digits_k10_f64_1797 - Electricity_k2_f8_45311
// MAX_POINTS 2048 (others) or 65536 (Electric) or 524288 (500K), 3145728, 14680064
// NF 2 (Test) 4 (Iris) 64 (Digits) 8 (Electricity)
// BLOCK_SIZE 128 (others) 64 (Digits)
// MAX_BLOCKS 256 (others) 1024 (Luna500K) 20048, 131072
// barcrawl_k10_f3_14057567 => 14680064 => 114688
#define MAX_POINTS 2621440
#define NF 20
#define BLOCK_SIZE 128

#define MAX_CLUSTERS 16
#define MAX_BLOCKS 20480


// Calculate the euclidian distance between two points in the CPU
float distance(float v1[], float v2[]) {
    float sum = 0.0;
    for (int d = 0; d < NF; d++) {
        sum += pow((v1[d] - v2[d]), 2);
    }
    return sqrt(sum);
}

// Calculate the euclidian distance between two points in the GPU
__device__ float distance(float *s_point, int p, int q, int nfeat) {
    float sum = 0.0;
    for (int d = 0; d < nfeat; d++) {
        sum = sum + pow((s_point[p+d] - s_point[q+d]), 2);
    }
    return sqrt(sum);
}

// Kernel that calculates the parcial sums (in each dimenstion) of the instances coordenates
__global__ void centroids(int cluster, float *d_centroid_tmp, float *d_point, int *d_cluster_start, int nfeat) {

  __shared__ float s_centroid[BLOCK_SIZE * NF];
  int tid = threadIdx.x;
  int lower = d_cluster_start[cluster];
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int size = d_cluster_start[cluster+1] - d_cluster_start[cluster];
  int p; //reduction step: 64, 32, 16, 8, 4, 2,1
    
  // All threads initialize with zero the shared memory
  for (int d = 0; d < nfeat; d++) {
    s_centroid[tid * nfeat + d] = 0.0;
  }
  __syncthreads();

  // Copy points from global memory to shared memory
  if (i < size) {
    for (int d = 0; d < nfeat; d++) {
      s_centroid[tid * nfeat + d] = (float ) d_point[(lower + i)*nfeat + d];
    }
  }
  __syncthreads();

  // Perform a local reduction on the memory shared data
  // It starts with 64 threads, then 32, 16, 8, 4, 2, 1
  p = blockDim.x / 2;
  while (p != 0) {
    if (tid < p) {
	    for (int d = 0; d < nfeat; d++) {
        s_centroid[tid*nfeat+d] = s_centroid[tid*nfeat+d] + s_centroid[(tid+p)*nfeat+d];
      }
    }
    __syncthreads();
    p = p/2;
  }

  // Thread zero of each block moves the local result to the global memory
  if (tid == 0) {
    for (int d = 0; d < nfeat; d++) {
        d_centroid_tmp[blockIdx.x * nfeat + d] = (float )s_centroid[d];
    }
  }
}

// Kernel that finds the cluster/point with the greatest distance from the centroid
//   maxs_intra<<<nblocks, BLOCK_SIZE>>>( cluster, d_index_tmp, d_maxs_tmp, d_centroid, d_point, d_cluster_start, nfeat );

__global__ void maxs_intra(int cluster, int *d_index_tmp, float *d_maxs_tmp, float *d_centroid, float *d_point, int *d_cluster_start, int nfeat) {
  
  __shared__ float s_point[(BLOCK_SIZE+1)*NF];
  __shared__ int s_pos[(BLOCK_SIZE+1)];
  __shared__ float s_dist[(BLOCK_SIZE+1)];
  int tid = threadIdx.x;
  int lower = d_cluster_start[cluster];
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  int size = d_cluster_start[cluster+1] - d_cluster_start[cluster];
  int nb, d, p, r, q;
  float dist;
  
  // All threads initialize shared memory
  // s_pos[tid] = lower + tid;
  s_pos[tid] = lower + i;
  s_dist[tid] = 0.0;
  for (d = 0; d < nfeat; d++) {
    s_point[tid*nfeat+d] = 0.0;
  }
  __syncthreads();

  // printf("\nNBlocks %d", gridDim.x);

  // Copy data points from global memory to shared memory
  if (i < size) {
    for (d = 0; d < nfeat; d++) {
      s_point[tid*nfeat+d] = d_point[(lower+i)*nfeat+d];
    }
  }
  __syncthreads();

  // store centroid in the last position of the vector shared memory to save memory
  if (tid == 0) {
    for (d = 0; d < nfeat; d++) {
      s_point[blockDim.x*nfeat+d] = d_centroid[cluster*nfeat+d]; 
    }       
  }
  __syncthreads();

  // adjust limit for the last block
  if (blockIdx.x == (gridDim.x -1)) {
    nb = size % blockDim.x;
  } else { 
    nb = blockDim.x; 
  }

  // each thread calculates dist
  if (tid < nb) {
    r = tid*nfeat; // point index
    q = blockDim.x*nfeat; // centroid index
    dist = distance(s_point, r, q, nfeat);
    // s_point[tid*nfeat] = dist;
    s_dist[tid] = dist;
  }
  __syncthreads();

  // reduction to find the maximum distance
  /* p = blockDim.x / 2; // log steps
  while (p != 0) {
    if (tid < p) {
      if (s_point[tid*nfeat] < s_point[(tid+p)*nfeat]) {
        s_point[tid*nfeat] = s_point[(tid+p)*nfeat];
        s_pos[tid] = s_pos[tid+p];  
      }
    }
    __syncthreads();
    p = p/2;
  } */

  p = blockDim.x / 2; // log steps
  while (p != 0) {
    if (tid < p) {
      if (s_dist[tid] < s_dist[tid+p]) {
        s_dist[tid] = s_dist[tid+p];
        s_pos[tid] = s_pos[tid+p];  
      }
    }
    __syncthreads();
    p = p/2;
  }

  // Thread zero of each block copy data to glocal memory
  if (tid == 0) {
    d_index_tmp[blockIdx.x] = s_pos[0];
    // d_maxs_tmp[blockIdx.x] = s_point[0];
    d_maxs_tmp[blockIdx.x] = s_dist[0];
  }
}

int main()
{
  int num_clusters; // number of clusters
  static int cluster_size[MAX_CLUSTERS]; //cluster sizes
  static float point[MAX_POINTS][NF]; // cluster data
  float *d_point; // GPU cluster data
  static float centroid[MAX_CLUSTERS+1][NF]; // centroid data
  float *d_centroid; // GPU centroid data
  static float centroid_tmp[MAX_BLOCKS][NF]; // centroid temporary data
  float *d_centroid_tmp; // GPU centroid temporary data
  static float centroid_global[NF]; // global centroid
  static int index_tmp[MAX_BLOCKS]; // index temporary data
  int *d_index_tmp; // GPU index temporary data
  static float maxs_tmp[MAX_BLOCKS]; // max values temporary
  float *d_maxs_tmp; // GPU max values temporary
  static int cluster_start[MAX_CLUSTERS+1]; // start cluster indexes
  int *d_cluster_start; // GPU start cluster indexes
  FILE *fp; // file pointer
  int size = 0; // total number of points
  int nfeat; // number of attributes
  clock_t start, stop; // measure time
  double running_time; // running time
  int nblocks; // number of blocks
  int cluster; // current cluster
  float sum; // sum of elements
  float dist; // distance
  double max_distance; // maximum distance
  float min_distance; // minimum distance
  int cluster1; // cluster chosen
  int p1; // index chosen

  // Input the number of clusters and the cluster information
  // Format: 1st line: #clusters #features, 2nd: cluster sizes, 3rd: data
 
  // fp = fopen("test_k2_f2_10.dat", "r");
  // fp = fopen("iris_k3_f4_150.dat", "r");
  // fp = fopen("digits_k12_f64_1797s.dat", "r");
  // fp = fopen("electricity_k2_f8_45311.dat", "r");
  // fp = fopen("iris_k3_f4_150.dat", "r");
  // fp = fopen("digits_k13_f64_1797s.dat", "r");
  // fp = fopen("luna_k9_f20_7000s.dat", "r");
  // fp = fopen("satimage_k8_f36_6430s.dat", "r");
  // fp = fopen("aggregation_k9_f2_788s.dat", "r");
   fp = fopen("/home/wellington/luna_files/luna_k5_f20_2500000.dat", "r");
  // fp = fopen("texture_k13_f40_5500s.dat", "r");
  // fp = fopen("barcrawl_k10_f3_14057567.dat", "r");

  // Read file (upload file first if running in Collab)
  fscanf(fp, "%d %d", &num_clusters, &nfeat);
  num_clusters = 8;
  for (int k = 0; k < num_clusters; k++) {
    fscanf(fp, "%d", &cluster_size[k]);
    size = size + cluster_size[k];
    //printf("\ncluster_size %d", cluster_size[k]);
  }
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < nfeat; j++) {
       // fscanf(fp, "%f", &point[i][j]);
       if ( fscanf(fp, "%f", &point[i][j]) == 1) {
         fscanf(fp, ",");
       }
    }
  } 
  fclose(fp);

  // prefix sum to find out the beginning of each cluster
  cluster_start[0] = 0;
  for (int i = 1; i < num_clusters+1; i++) {
    cluster_start[i] = cluster_start[i-1] + cluster_size[i-1];
  }

  // Allocate GPU memory
  hipMalloc(&d_cluster_start, (MAX_CLUSTERS+1)*sizeof(int));
  hipMalloc(&d_point, MAX_POINTS*NF*sizeof(float));
  hipMalloc(&d_centroid_tmp, MAX_BLOCKS*NF*sizeof(float));
  hipMalloc(&d_index_tmp, MAX_BLOCKS*sizeof(int));
  hipMalloc(&d_maxs_tmp, MAX_BLOCKS*sizeof(float));
  hipMalloc(&d_centroid, MAX_CLUSTERS*NF*sizeof(float));

  // start clock to measure running time
  start = clock();

  // Copy data (cluster points and start indices) to the GPU
  hipMemcpy(d_point, point, MAX_POINTS*nfeat*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_cluster_start, cluster_start, (MAX_CLUSTERS+1)*sizeof(int), hipMemcpyHostToDevice);

  // find centroids: launch the kernel for each cluster
  for (cluster = 0; cluster < num_clusters; cluster++) {
    
    // Number of blocks is size of cluster divided by the block size
    nblocks = (cluster_size[cluster] + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // launch kernel and verify if got any error
    centroids<<<nblocks, BLOCK_SIZE>>>( cluster, d_centroid_tmp, d_point, d_cluster_start, nfeat );
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) { printf("CUDA error: %s\n", hipGetErrorString(error)); exit(-1); }

    // Wait for the kernel to finish and copy centroid temporary data for the host (CPU)
    // The kernel returns the parcial sums of each block
    hipDeviceSynchronize();
    hipMemcpy(&centroid_tmp, d_centroid_tmp, MAX_BLOCKS*NF*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Calculate centroid and store it in the centroid_tmp
    // The parcial sums need to be accumulated and divided by the cluster size
    for (int i = 0; i < nfeat; i++) {
      sum = 0.0;
      for (int j = 0; j < nblocks; j++) {
        sum = sum + centroid_tmp[j][i];
      }
      centroid_global[i] = centroid_global[i] + sum;
      centroid_tmp[0][i] = sum / (float )cluster_size[cluster];
      centroid[cluster][i] = centroid_tmp[0][i];
    } 
  }

  //printf("\nCentroid Global: ");
  for (int i = 0; i < nfeat; i++) {
    centroid_global[i] = centroid_global[i] / size;
  }

  // Copy centroids to the GPU
  hipMemcpy(d_centroid, centroid, MAX_CLUSTERS*NF*sizeof(float), hipMemcpyHostToDevice);

  // Find the centroid closer to the global centroid
  min_distance = DBL_MAX;
  for (int i = 0; i < num_clusters; i++) {
    dist = distance(centroid[i], centroid_global);                               
    if (dist < min_distance) {
      min_distance = dist;
    }
  }
  // Now min_distance is the numerator of the Dunn index

  // Now, find maximum radius launching the kernel for each cluster again
  max_distance = 0.0;
  for (cluster = 0; cluster < num_clusters; cluster++) {
    
    // Number of blocks is size of cluster divided by the block size
    nblocks = (cluster_size[cluster] + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    // launch kernel and verify if got any error
    maxs_intra<<<nblocks, BLOCK_SIZE>>>( cluster, d_index_tmp, d_maxs_tmp, d_centroid, d_point, d_cluster_start, nfeat );
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) { printf("CUDA error: %s\n", hipGetErrorString(error)); exit(-1); }

    // Wait for the kernel to finish and copy maximum temporary data for the host (CPU)
    // The kernel returns the several maximums, one for each block
    hipDeviceSynchronize();
    hipMemcpy(&maxs_tmp, d_maxs_tmp, MAX_BLOCKS*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&index_tmp, d_index_tmp, MAX_BLOCKS*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Calculate the global maximum and store it in the max_distance
    // The parcial maximums need to be compared and the global maximum stored
    // The cluster and the maximum point position need to be saved (cluster1 and p1)
    for (int j = 0; j < nblocks; j++) {
      if (maxs_tmp[j] > max_distance) {
        max_distance = maxs_tmp[j];
        p1 = index_tmp[j];
        cluster1 = cluster;
      }
    }
  }

  printf("\nCluster1: %d, p1: %d, max_distance: %.2f", cluster1, p1, max_distance);
/*
  // Now that we know the cluster (cluster1) with the point (p1) furthest to the 
  // centroid, we can calculate de diameter as the maximum distance between p1 
  // and another point in the same cluster
 
  // store p1 in the centroid[cluster1] vector, to re-use space, and move it to the GPU
  for (int d = 0; d < nfeat; d++) {
    // centroid[0][d] = point[p1][d];
    centroid[cluster1][d] = point[p1][d];
  } 
  hipMemcpy(d_centroid, centroid, MAX_CLUSTERS*NF*sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  
  // Find maximum diameter in the right cluster (cluster1)
  // The point p1 is compared to all points of cluster1
  // This is done lauching the max_intra kernel once more
  cluster = cluster1;
  nblocks = (cluster_size[cluster] + (BLOCK_SIZE - 1)) / (BLOCK_SIZE);

  maxs_intra<<<nblocks, BLOCK_SIZE>>>( cluster, d_index_tmp, d_maxs_tmp, d_centroid, d_point, d_cluster_start, nfeat );
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)  { printf("CUDA error: %s\n", hipGetErrorString(error)); exit(-1); }

  // Wait for the kernel to finish and copy maximum temporary data for the host (CPU)
  // The kernel returns the several maximums, one for each block
  hipDeviceSynchronize();
  hipMemcpy(&maxs_tmp, d_maxs_tmp, MAX_BLOCKS*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&index_tmp, d_index_tmp, MAX_BLOCKS*sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Find out the maximum distance (one per block)
  // This is the denominator of the Dunn index
  max_distance = 0.0;
  for (int j = 0; j < nblocks; j++) {
    if (maxs_tmp[j] > max_distance) {
      max_distance = maxs_tmp[j];
    }
  }
*/
  // finalize runtime calculation
  stop = clock();
   
  // Print results
  printf("\nMin intercluster %.2f", min_distance);
  printf("\nMax intracluster %.2f", max_distance);
  printf("\nThe Dunn index: %.4f", min_distance / max_distance);
  
  // Print the time taken
  running_time = (double)(stop - start) / CLOCKS_PER_SEC;
  printf("\nTime taken: %lf milissegundos\n", 1000.0*running_time);

  // Free GPU memory
  hipFree( d_cluster_start );
  hipFree( d_point );
  hipFree( d_centroid_tmp );
  hipFree( d_index_tmp );
  hipFree( d_maxs_tmp );
  hipFree( d_centroid );

  return 0;
}
